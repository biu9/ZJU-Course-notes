#include <hip/hip_runtime.h>

#include <iomanip>
#include <iostream>
#include <random>

const int kSize = 5000;
const int kKernelSize = 13;  // odd

#define InitRandom()                         \
  std::random_device r;                      \
  std::default_random_engine generator(r()); \
  std::uniform_real_distribution<float> distribution(0, 1e3);

void Generate(float *const a, float *const w) {
#pragma omp parallel for num_threads(8)
  for (int i = 0; i < kSize; ++i) {
    InitRandom();
    const int j_upperbound = (i + 1) * kSize; 
    for (int j = i * kSize; j < j_upperbound; ++j)
      a[j] = distribution(generator);
  }
  {
    InitRandom();
    for (int i = 0; i < kKernelSize * kKernelSize; ++i)
      w[i] = distribution(generator);
  }
}

__global__
void Conv_GPU(const float *const a, const float *const w, float *const b) {
#pragma omp parallel for num_threads(8)
  //for (int i = 0; i < kSize; ++i) {
    //for (int j = 0; j < kSize; ++j) {
      int index_i = blockIdx.x * blockDim.x + threadIdx.x;
      int index_j = blockIdx.y * blockDim.y + threadIdx.y;
//      int shared_index = threadIdx.y + threadIdx.x * 16;

int gridStride = blockDim.x * gridDim.x;

__shared__ float b_temp[256];

for(int i = index_i;i<kSize;i+=gridStride){
for(int j = index_j;j<kSize;j+=gridStride){
  if(i < kSize && j < kSize){
    b[i * kSize + j] = 0;
    int x = i - kKernelSize / 2;
    int y = j - kKernelSize / 2;

    for (int k = 0; k < kKernelSize; ++k) {
      for (int l = 0; l < kKernelSize; ++l) {
        if (!(x < 0 || x >= kSize || y < 0 || y >= kSize)){
        b_temp[threadIdx.x*16 + threadIdx.y] += a[x * kSize + y] * w[k * kKernelSize + l];
//            __syncthreads();
//            b[i * kSize + j] = b_temp[index_i * 64 + index_j];
//              b_temp[i * kSize + j] += a[x * kSize + y] * w[k * kKernelSize + l];
//              b[i * kSize + j] += a[x * kSize + y] * w[k * kKernelSize + l];
//            b[i * kSize + j] = b_temp[(blockIdx.x * blockDim.x + threadIdx.x)*32+blockIdx.y * blockDim.y + threadIdx.y];
        }
        y++;
      }
      x++;
      y -= kKernelSize;
    }
//        __syncthreads();
   b[i * kSize + j] = b_temp[threadIdx.x*16 + threadIdx.y];
   b_temp[threadIdx.x*16 + threadIdx.y] = 0;
//       __syncthreads();
  }
}
}
}

void Conv(const float *const a, const float *const w, float *const b) {
  #pragma omp parallel for num_threads(8)
    for (int i = 0; i < kSize; ++i) {
      for (int j = 0; j < kSize; ++j) {
        float conv = 0;
        int x = i - kKernelSize / 2, y = j - kKernelSize / 2;
        for (int k = 0; k < kKernelSize; ++k) {
          for (int l = 0; l < kKernelSize; ++l) {
            if (!(x < 0 || x >= kSize || y < 0 || y >= kSize))
              conv += a[x * kSize + y] * w[k * kKernelSize + l];
            y++;
          }
          x++;
          y -= kKernelSize;
        }
        b[i * kSize + j] = conv;
      }
    }
  }

void Check(const float *const a, const float *const w, float *const b) {
  auto b_std = new float[kSize * kSize];
  Conv(a, w, b_std);
  for (int i = 0; i < kSize * kSize; ++i) {
    if (abs(b[i] / b_std[i] - 1) > 1e-3 || isnanf(b[i]) || isinff(b[i])) {
      std::cout << "\x1b[31m"
                   "Wrong Answer"
                   "\x1b[0m"
                   " at "
                << i << std::endl;
      std::cout << "expected " << b_std[i] << " but found " << b[i]
                << std::endl;
      delete[] b_std;
      return;
    }
}
std::cout << "\x1b[32m"
             "Correct"
             "\x1b[0m"
          << std::endl;

delete[] b_std;
}

void Output(const float *const a, const float *const w, const float *const b) {
for (int i = 0; i < kSize; ++i) {
  for (int j = 0; j < kSize; ++j)
    std::cout << std::setw(2) << a[i * kSize + j] << ' ';
  std::cout << std::endl;
}

for (int i = 0; i < kKernelSize; ++i) {
  for (int j = 0; j < kKernelSize; ++j)
    std::cout << std::setw(2) << w[i * kKernelSize + j] << ' ';
  std::cout << std::endl;
}

for (int i = 0; i < kSize; ++i) {
  for (int j = 0; j < kSize; ++j)
    std::cout << std::setw(2) << b[i * kSize + j] << ' ';
  std::cout << std::endl;
}
}

int main() {
    auto a = new float[kSize * kSize];
    auto w = new float[kKernelSize * kKernelSize];
    auto b = new float[kSize * kSize];
  
    float *a_cuda;
    float *b_cuda;
    float *w_cuda;
  
  //  auto a_cuda = new float[kSize * kSize];
  //  auto w_cuda = new float[kKernelSize * kKernelSize];
  //  auto b_cuda = new float[kSize * kSize];
  
    size_t size = kSize * kSize * sizeof (float);
    size_t size_w = kKernelSize * kKernelSize * sizeof (float);
  
    hipMalloc((void**)&a_cuda,size);
    hipMalloc((void**)&b_cuda,size);
    hipMalloc((void**)&w_cuda,size_w);
  //  cudaMallocManaged (&a, size);//===========================
  //  cudaMallocManaged (&b, size);//===========================
  //  cudaMallocManaged (&w, size_w);//=========================
  
    Generate(a, w);
  
    hipEvent_t start_e, stop_e;
    hipEventCreate(&start_e);
    hipEventCreate(&stop_e);
  
    hipEventRecord(start_e);
  
    dim3 threads_per_block (8, 8, 1); // A 16 x 16 block threads==========================
    dim3 number_of_blocks (256, 256, 1);//=============================
  
    hipMemcpy(a_cuda,a,size,hipMemcpyHostToDevice);
  //  cudaMemcpy((void*)b_cuda,(void*)b,size,cudaMemcpyHostToDevice);
    hipMemcpy(w_cuda,w,size_w,hipMemcpyHostToDevice);
  
      hipStream_t stream;//------------------------------------------------------
      hipStreamCreate(&stream);//-------------------------------------------------
  
    Conv_GPU <<<number_of_blocks,threads_per_block,0,stream>>> (a_cuda, w_cuda, b_cuda);//================================
  
      hipStreamDestroy(stream);//-------------------------------------------------
  
  //  cudaMemcpy((void*)a,(void*)a_cuda,size,cudaMemcpyDeviceToHost);
    hipMemcpy(b,b_cuda,size,hipMemcpyDeviceToHost);
  //  cudaMemcpy((void*)w,(void*)w_cuda,size_w,cudaMemcpyDeviceToHost);
  
    hipDeviceSynchronize();
  
    hipEventRecord(stop_e);
    hipEventSynchronize(stop_e);
  
    Check(a, w, b);
  
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_e, stop_e);
    std::cout << milliseconds << " milliseconds" << std::endl;
    hipEventDestroy(start_e);
    hipEventDestroy(stop_e);
  
    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(w_cuda);
  
  //  cudaFree(a); //============================
  //  cudaFree(b);//=============================
  //  cudaFree(w);//=============================
  
    // Output(a, w, b);
  
    delete[] a;
    delete[] w;
    delete[] b;
    return 0;
  }
  
 